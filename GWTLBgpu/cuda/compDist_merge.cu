#include "hip/hip_runtime.h"
/*
compDist_merge.cu

comparing distributions organized into the rows of two matrices A and B.
This is part of the computation of the Third Lower Bound (TLB) of network Gromov-Wasserstain distance 
per the work of Chowdhury & Memoli (2019).

This is a parallelization of part of the function "tools/compareRealDistibutions.m" in the repo (https://github.com/samirchowdhury/GWnets).
Here we assume, however, the inputs are already sorted, i.e. A, B, and the corresponding cumulative probability measures cmA, cmB. 

Compare to an earlier version of this program "compDist_mf.cu" (9/6/2019), the present version use a more efficient merge before sorting. 

This is not designed to compare matrices greater than 2000x2000. 
More precisely, 2 + 4 * (NA + NB) * sizeof(float) should not exceed 48k.
I.e. NA + NB <= 3071.

== to compile in MATLAB
mexcuda compDist_merge.cu
----------------
created by Mengsen Zhang, mengsenzhang@gmail.com (9/18/2019)

*/
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <iostream>
#include <iomanip>

#include "mex.h"

using namespace std;

// #define UNIQUETOL 1e-10 // uncomment if you want to add a "unique" procedure while computing the joint distribution; also uncomment corresponding lines in "mergesort".
#define N_REGS_PER_THREAD 52 // remember to update this if the program is change!

hipError_t compdist(const double* A, const double* B, const double* cmA, const double* cmB, const unsigned int NA, const unsigned int NB, double* dist);
unsigned int nextPow2(unsigned int n);
__global__ void compdistKernel(const double* A, const double* B, const double* CMA, const double* CMB, const unsigned int NA, const unsigned int NB, double* dist);
__device__ void mergesort(const float* x, const float* y, float* xy, int* zeroflag, const int nx, const int ny);
__device__ int lastPow2(int n);
__device__ void findIdx(const float* cm, const float* CM, int* idx, const int N_cm, const int N_CM);
template<typename num> __device__ num sum(num* x, const int len);
template<typename num> __device__ void reset(num* x, const int len);

void mexFunction(int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[]) {
	/*
	Interface with matlab, need 6 inputs, and 1 output.
	*/
	if (nlhs != 1) {
		mexErrMsgIdAndTxt("mexFun:nlhs", "need 1 output: dist");
	}
	if (nrhs != 6) {
		mexErrMsgIdAndTxt("mexFun:nrhs", "need 6 inputs: sorted_A, sorted_B, sorted_cmA, sorted_cmB, NA, NB");
	}

	// read input
	double *A = mxGetPr(prhs[0]);
	double *B = mxGetPr(prhs[1]);
	double *cmA = mxGetPr(prhs[2]);
	double *cmB = mxGetPr(prhs[3]);
	int NA = mxGetScalar(prhs[4]);
	int NB = mxGetScalar(prhs[5]);

	// prep output
	plhs[0] = mxCreateDoubleMatrix(NB, NA, mxREAL);
	double *dist = mxGetPr(plhs[0]);

	// compute!
	hipError_t cudaStatus = compdist(A, B, cmA, cmB, (unsigned int) NA, (unsigned int) NB, dist);
	if (cudaStatus != hipSuccess) {
		mexErrMsgIdAndTxt("compdist:compdist", "main function didn't run correctly!");
	}
}

// host function for launching the kernel
hipError_t compdist(const double* A, const double* B, const double* cmA, const double* cmB, 
					 const unsigned int NA, const unsigned int NB, 
					 double* dist) {
	const unsigned int N_max = NA + NB + 1;
	// device data points
	double *dev_cmA, *dev_cmB, *dev_A, *dev_B, *dev_dist;

	// check device
	hipError_t cudaStatus = hipSuccess;
	hipDeviceReset();
	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, 0);
	
	// declare variables
	int count;
	size_t requiredsharemem;
	size_t dyn_sharedmemsize;
	const unsigned int max_regs_threads = (prop.regsPerBlock-1)/N_REGS_PER_THREAD + 1;//max # threads per block limited by # registers
	const unsigned int N_optim_threads = min(N_max, prop.warpSize*prop.multiProcessorCount);//optimal threads per block for all MP to be working
	const unsigned int blocksize = min(N_optim_threads, min(max_regs_threads,prop.maxThreadsPerBlock));
	// mexPrintf("blocksize=%d\n", blocksize);
	dim3 griddims(NA, NB, 1);
	dim3 blockdims(blocksize, 1, 1);

	cudaStatus = hipGetDeviceCount(&count);
	if (cudaStatus != hipSuccess) {
		mexPrintf("Number of device: %d\n", count);
		mexErrMsgIdAndTxt("compDist:devicecount ", "Could not find device.!");
		goto Error;
	}

	// check shared memory size
	requiredsharemem = 2 + 4 * (NA + NB) * sizeof(float);
	
	hipDeviceSetCacheConfig(hipFuncCachePreferShared);
	dyn_sharedmemsize = prop.sharedMemPerBlock;
	if (requiredsharemem > dyn_sharedmemsize) {
		mexPrintf("Requested shared memory %d bytes > available %d bytes.\n", requiredsharemem, dyn_sharedmemsize);
		mexWarnMsgTxt("The shared memory required for your job may exceed the capacity of your GPU.");
	}

	// choose device
	if (count>0) {
		cudaStatus = hipSetDevice(0);
		if (cudaStatus != hipSuccess) {
			mexErrMsgIdAndTxt("compdist:hipSetDevice", "cannot set device 0!\n");
			goto Error;
		}
	}

	// allocate memory at device
	cudaStatus = hipMalloc((void**)&dev_cmA, NA * NA * sizeof(double));
	if (cudaStatus != hipSuccess) {
		mexErrMsgIdAndTxt("compdist:hipMalloc", "Can't Malloc dev_cmA!\n");
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&dev_cmB, NB * NB * sizeof(double));
	if (cudaStatus != hipSuccess) {
		mexErrMsgIdAndTxt("compdist:hipMalloc", "Can't Malloc dev_cmB!\n");
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&dev_A, NA * NA * sizeof(double));
	if (cudaStatus != hipSuccess) {
		mexErrMsgIdAndTxt("compdist:hipMalloc", "Can't Malloc dev_A!\n");
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&dev_B, NB * NB * sizeof(double));
	if (cudaStatus != hipSuccess) {
		mexErrMsgIdAndTxt("compdist:hipMalloc", "Can't Malloc dev_B!\n");
		goto Error;
	} 
	cudaStatus = hipMalloc((void**)&dev_dist, NA * NB * sizeof(double));
	if (cudaStatus != hipSuccess) {
		mexErrMsgIdAndTxt("compdist:hipMalloc", "Can't Malloc dev_dist!\n");
		goto Error;
	} 
    cudaStatus = hipMemset(dev_dist, 99.0, NA * NB * sizeof(double));
    if (cudaStatus != hipSuccess) {
		mexErrMsgIdAndTxt("compdist:hipMalloc", "Can't Memset dev_dist!\n");
		goto Error;
	}
	// transfer data
	cudaStatus = hipMemcpy(dev_cmA, cmA, NA * NA * sizeof(double), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		mexErrMsgIdAndTxt("compdist:hipMemcpyHostToDevice", "Can't transfer cmA to device!\n");
		goto Error;
	}
	cudaStatus = hipMemcpy(dev_cmB, cmB, NB * NB * sizeof(double), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		mexErrMsgIdAndTxt("compdist:hipMemcpyHostToDevice", "Can't transfer cmB to device!\n");
		goto Error;
	}
	cudaStatus = hipMemcpy(dev_A, A, NA * NA * sizeof(double), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		mexErrMsgIdAndTxt("compdist:hipMemcpyHostToDevice", "Can't transfer A to device!\n");
		goto Error;
	}
	cudaStatus = hipMemcpy(dev_B, B, NB * NB * sizeof(double), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		mexErrMsgIdAndTxt("compdist:hipMemcpyHostToDevice", "Can't transfer A to device!\n");
		goto Error;
	}

	// launch kernel	

	compdistKernel<<<griddims, blockdims, min(requiredsharemem,dyn_sharedmemsize)>>>(dev_A, dev_B, dev_cmA, dev_cmB, NA, NB, dev_dist);

	// check for errors
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		mexPrintf("hipGetLastError returned error code %d after launching compdistKernel!\n", cudaStatus);
		mexErrMsgIdAndTxt("compdist:hipGetLastError", "kernel didn't run correctly!\n");
		goto Error;
	}
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		mexPrintf("hipDeviceSynchronize returned error code %d after launching compdistKernel!\n", cudaStatus);
		mexErrMsgIdAndTxt("compdist:hipDeviceSynchronize", "device didn't sync!\n");
		goto Error;
	}

	// copy results to host
	cudaStatus = hipMemcpy(dist, dev_dist, NA * NB * sizeof(double), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		mexErrMsgIdAndTxt("compdist:hipMemcpyDeviceToHost", "couldn't get data back!\n");
		goto Error;
	}

	hipDeviceReset();
Error:
	hipDeviceReset();
	return cudaStatus;
}

unsigned int nextPow2(unsigned int n) {
	n--;
	n |= n >> 1;
	n |= n >> 2;
	n |= n >> 4;
	n |= n >> 8;
	n |= n >> 16;
	n++;
	return n;
}

__global__ void compdistKernel(const double* A, const double* B, 
								const double* CMA, const double* CMB,
								const unsigned int NA, const unsigned int NB,
								double* dist) {
	// basic info
	int idx_vA0 = blockIdx.x*NA; // start of the ii-th row of A
	int idx_vB0 = blockIdx.y*NB; // start of the jj-th row of B
	int N_merged = NA + NB + 1;

	// prepare shared memory
	extern __shared__ float ins[];
	float *cmA = ins;
	float *cmB = (float*)&cmA[NA]; // # in brack is the size of the previous variable!
	float *cmMerged = (float*)&cmB[NB];
	int *zeroflag = (int*)&cmMerged[N_merged];
	float *diffAB = (float*)&zeroflag[N_merged];

	// load cdf's from global memory
	int N_steps = (NA - 1) / blockDim.x + 1;// for large vectors, we load in steps
	for (int n = 0; n < N_steps; n++) {
		int idx_v = n*blockDim.x + threadIdx.x;
		int idx_A = idx_vA0 + idx_v;
		if (idx_v < NA) {
			cmA[idx_v] = __double2float_rn(CMA[idx_A]);
		}
	}
	__syncthreads();
	N_steps = (NB - 1) / blockDim.x + 1;// for large vectors, we load in steps
	for (int n = 0; n < N_steps; n++) {
		int idx_v = n*blockDim.x + threadIdx.x;
		int idx_B = idx_vB0 + idx_v;
		if (idx_v < NB) {
			cmB[idx_v] = __double2float_rn(CMB[idx_B]);
		}
	}
	__syncthreads();

	// merge and sort cdf's
	reset<int>(zeroflag,N_merged);
	reset<float>(cmMerged,N_merged);
	__syncthreads();
	if (NA>NB) {
		mergesort(cmA, cmB, cmMerged, zeroflag, NA, NB);// the algorithm assume the first argument is larger
	} else{
		mergesort(cmB, cmA, cmMerged, zeroflag, NB, NA);
	}
	
	__syncthreads();

	int N_dCM = N_merged - 1;
	reset<int>(zeroflag,N_merged);
	// finding indices for vA (reuse "zeroflag" to store indices)
	findIdx(cmA, cmMerged, zeroflag, NA, N_dCM);
	// load A by idx_vA
	N_steps = (N_dCM - 1) / blockDim.x + 1;
	for (int n = 0; n < N_steps; n++) {
		int idx_idx_vA = n*blockDim.x + threadIdx.x;
		if (idx_idx_vA < N_dCM) {
			diffAB[idx_idx_vA] = __double2float_rn(A[idx_vA0 + zeroflag[idx_idx_vA]]);// reuse cmA as vA
		}
	}
	__syncthreads();

	// finding indices for vB
	findIdx(cmB, cmMerged, zeroflag, NB, N_dCM);

	// load B by idx_vB and subtract A
	for (int n = 0; n < N_steps; n++) {
		int idx_idx_vB = n*blockDim.x + threadIdx.x;
		if (idx_idx_vB < N_dCM) {
			diffAB[idx_idx_vB] = powf(diffAB[idx_idx_vB] - __double2float_rn(B[idx_vB0 + zeroflag[idx_idx_vB]]), 2)*(cmMerged[idx_idx_vB + 1] - cmMerged[idx_idx_vB]);
		}
	}
	__syncthreads();

	// output final distance
	double dist_ij = (double) sum<float>(diffAB, N_dCM); 
	__syncthreads();
	if (threadIdx.x == 0) {
		dist[blockIdx.x*NB + blockIdx.y] = sqrtf(dist_ij);//cmMerged[(blockIdx.x*NB + blockIdx.y)%N_merged];//
	}
	__syncthreads();
}

__device__ void findIdx(const float* cm, const float* CM, int* idx, const int N_cm, const int N_CM) {
	reset<int>(idx, N_CM);
	int N_steps = (N_CM - 1) / blockDim.x + 1;
	// for each cm : compare to CM values in parallel
	for (int n = 0; n < N_steps; n++) {
		int idx_CM = n*blockDim.x + threadIdx.x;
		if ( idx_CM < N_CM ) {
			for (int m = 0; m < N_cm; m++) {// loop through values of cm
				idx[idx_CM] = min(idx[idx_CM] + (cm[m] <= CM[idx_CM]), N_cm - 1);//count the number of cm <=CM_i
				__syncthreads();
			}
		}
		__syncthreads();
	}	
	__syncthreads();
}

__device__ void mergesort(const float* x, const float* y,
	float* xy, int* zeroflag,
	const int nx, const int ny) {
	int N = nx + ny; //max N
	int Nsteps = (N - 1) / blockDim.x + 1;
	// -- merge: interleaving two vectors assuming nx>ny
	// distribute x over a NA+NB long vector
	for (int n = 0; n<Nsteps; n++) {
		int idx_x = n*blockDim.x + threadIdx.x;
		if (idx_x < nx) {
			xy[(int) (idx_x*N)/nx + 1] = x[idx_x];
		}
	}
	__syncthreads();
	// distribute y between x
	for (int n = 0; n<Nsteps; n++) {
		int idx_xy = n*blockDim.x + threadIdx.x;
		if ((idx_xy + 1 < N) && (xy[idx_xy + 2] == 0) ) {// if there is a gap to the right of idx_xy
			xy[idx_xy + 2] = y[idx_xy*ny/N];//insert y
		}
	}
	__syncthreads();

	// sort
	int N_comp = (N + 1) / 2;// max number of comparisons per loop
	Nsteps = (N_comp - 1) / blockDim.x + 1;
	int change;
	do {
		// use zeroflag to keep track of change
		reset<int>(zeroflag, N_comp);
		change = 0;
		__syncthreads();

		// comparing neighboring points
		for (int n = 0; n<Nsteps; n++) {
			int idxcomp = n*blockDim.x + threadIdx.x;			
			if (idxcomp * 2 + 1 < N + 1) {
				// compare : round 1
				float left = xy[idxcomp * 2];
				float right = xy[idxcomp * 2 + 1];
				// bool sim = (abs(left - right) < UNIQUETOL);
				xy[idxcomp * 2] = min(left, right);// * (float) (!sim);// zero if two numbers are similar
				xy[idxcomp * 2 + 1] = max(left, right);// * (float) (!(sim && (min(left, right) == 0)));
				zeroflag[idxcomp] += (left != xy[idxcomp * 2]);// || (right != xy[idxcomp * 2 + 1]);
			}
		}
		__syncthreads();
		for (int n = 0; n<Nsteps; n++) {
			int idxcomp = n*blockDim.x + threadIdx.x;
			if (idxcomp * 2 + 2 < N + 1) {
				// round 2
				float left = xy[idxcomp * 2 + 1];
				float right = xy[idxcomp * 2 + 2];
				// bool sim = (abs(left - right) < UNIQUETOL);
				xy[idxcomp * 2 + 1] = min(left, right);// * (float) (!sim);// zero if two numbers are similar
				xy[idxcomp * 2 + 2] = max(left, right);// * (float) (!(sim && (min(left, right) == 0)));
				zeroflag[idxcomp] += (left != xy[idxcomp * 2 + 1]);// || (right != xy[idxcomp * 2 + 2]);
			}
		}
		__syncthreads();
		change = sum<int>(zeroflag, N_comp);
		__syncthreads();
	} while (change);
}

// the largest power of 2 smaller than input
__device__ int lastPow2(int n) {
	// next power of 2
	n--;
	n |= n >> 1;
	n |= n >> 2;
	n |= n >> 4;
	n |= n >> 8;
	n |= n >> 16;
	n++;
	// last power of 2
	return n >> 1;
}

// sum elements along x within a block
template<typename num>
__device__ num sum(num* x, const int len) {
	int n_sums0 = lastPow2(len); // max number of cols in the initial sum step
	for (int n_sums = n_sums0; n_sums > 0; n_sums >>= 1) {
		int N_steps = (n_sums - 1) / blockDim.x + 1;
		for (int n = 0; n<N_steps; n++) {
			int idx = n*blockDim.x + threadIdx.x;
			if (idx < n_sums && (idx + n_sums) < len) {
				x[idx] += x[idx + n_sums];
			}
			__syncthreads();
		}
		__syncthreads();
	}
	__syncthreads();
	return x[0];
}

// reset all elements to zero
template<typename num>
__device__ void reset(num* x, const int len) {
	int N_steps = (len - 1) / blockDim.x + 1;
	for (int n = 0; n < N_steps; n++) {
		int idx = n*blockDim.x + threadIdx.x;
		if (idx < len) {
			x[idx] = 0;
		}
		__syncthreads();
	}
}
