#include "hip/hip_runtime.h"
/*
compDist_uni.cu

comparing distributions organized into the rows of two matrices A and B.
This is part of the computation of the Third Lower Bound (TLB) of network Gromov-Wasserstain distance 
per the work of Chowdhury & Memoli (2019).

Here we assume probability distributions mA and mB are uniform. This greatly simplify the computation. 


== to compile in MATLAB
mexcuda compDist_uni.cu

== warning
this is not designed to handle comparisons greater than 4000*4000;
----------------
created by Mengsen Zhang, mengsenzhang@gmail.com (9/8/2019).

*/
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <iostream>
#include <iomanip>

#include "mex.h"

using namespace std;

#define N_THREADS_PER_BLOCK 1024

hipError_t compdist(const double* A, const double* B, const double* idxA, const double* idxB, const double* dCM,
					 const unsigned int NA, const unsigned int NB, const unsigned int NdCM,
					 double* dist);
__global__ void compdistKernel(const double* A, const double* B, 
								const double* idxA, const double* idxB, const double* dCM,
								const unsigned int NA, const unsigned int NB, const unsigned int NdCM,
								double* dist);
__device__ float sum(float* x, const int len);
__device__ int lastPow2(int n);



void mexFunction(int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[]) {
	/*
	Interface with matlab, need 6 inputs, and 1 output.
	*/
	if (nlhs != 1) {
		mexErrMsgIdAndTxt("mexFun:nlhs", "need 1 output: dist");
	}
	if (nrhs != 8) {
		mexErrMsgIdAndTxt("mexFun:nrhs", "need 8 inputs: sorted_A, sorted_B, idxA, idxB, dCM, NA, NB, NdCM");
	}

	// read input
	double *A = mxGetPr(prhs[0]);
	double *B = mxGetPr(prhs[1]);
	double *idxA = mxGetPr(prhs[2]);
	double *idxB = mxGetPr(prhs[3]);
	double *dCM = mxGetPr(prhs[4]);
	int NA = mxGetScalar(prhs[5]);
	int NB = mxGetScalar(prhs[6]);
	int NdCM = mxGetScalar(prhs[7]);

	// prep output
	plhs[0] = mxCreateDoubleMatrix(NB, NA, mxREAL);
	double *dist = mxGetPr(plhs[0]);

	// compute!
	hipError_t cudaStatus = compdist(A, B, idxA, idxB, dCM, (unsigned int) NA, (unsigned int) NB, (unsigned int) NdCM, dist);
	if (cudaStatus != hipSuccess) {
		mexErrMsgIdAndTxt("compdist:compdist", "main function didn't run correctly!");
	}
}

// host function for launching the kernel
hipError_t compdist(const double* A, const double* B, const double* idxA, const double* idxB, const double* dCM,
					 const unsigned int NA, const unsigned int NB, const unsigned int NdCM,
					 double* dist) {
	
	// device data points
	double *dev_A, *dev_B, *dev_dist, *dev_idxA, *dev_idxB, *dev_dCM;

	// check device
	hipError_t cudaStatus = hipSuccess;
	hipDeviceReset();
	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, 0);
	
	// declare variables
	int count;
	size_t requiredsharemem;
	size_t dyn_sharedmemsize;
	const unsigned int blocksize = min(NdCM, N_THREADS_PER_BLOCK);
	// size_t smsize = (size_t) min(requiredsharemem,dyn_sharedmemsize);
	dim3 griddims(NA, NB, 1);
	dim3 blockdims(blocksize, 1, 1);

	cudaStatus = hipGetDeviceCount(&count);
	if (cudaStatus != hipSuccess) {
		mexPrintf("Number of device: %d\n", count);
		mexErrMsgIdAndTxt("compDist:devicecount ", "Could not find device.!");
		goto Error;
	}

	// check shared memory size
	requiredsharemem = (NA + NB + NdCM) * sizeof(float);
	
	dyn_sharedmemsize = prop.sharedMemPerBlock;
	if (requiredsharemem > dyn_sharedmemsize) {
		mexPrintf("Requested shared memory %d bytes > available %d bytes.\n", requiredsharemem, dyn_sharedmemsize);
		mexWarnMsgTxt("The shared memory required for your job may exceed the capacity of your GPU.");
	}

	// choose device
	if (count>0) {
		cudaStatus = hipSetDevice(0);
		if (cudaStatus != hipSuccess) {
			mexErrMsgIdAndTxt("compdist:hipSetDevice", "cannot set device 0!\n");
			goto Error;
		}
	}

	// allocate memory at device
	cudaStatus = hipMalloc((void**)&dev_A, NA * NA * sizeof(double));
	if (cudaStatus != hipSuccess) {
		mexErrMsgIdAndTxt("compdist:hipMalloc", "Can't Malloc dev_A!\n");
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&dev_B, NB * NB * sizeof(double));
	if (cudaStatus != hipSuccess) {
		mexErrMsgIdAndTxt("compdist:hipMalloc", "Can't Malloc dev_B!\n");
		goto Error;
	} 
	cudaStatus = hipMalloc((void**)&dev_dist, NA * NB * sizeof(double));
	if (cudaStatus != hipSuccess) {
		mexErrMsgIdAndTxt("compdist:hipMalloc", "Can't Malloc dev_dist!\n");
		goto Error;
	} 

	cudaStatus = hipMalloc((void**)&dev_idxA, NdCM * sizeof(double));
	if (cudaStatus != hipSuccess) {
		mexErrMsgIdAndTxt("compdist:hipMalloc", "Can't Malloc dev_cmA!\n");
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&dev_idxB, NdCM * sizeof(double));
	if (cudaStatus != hipSuccess) {
		mexErrMsgIdAndTxt("compdist:hipMalloc", "Can't Malloc dev_cmB!\n");
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&dev_dCM, NdCM * sizeof(double));
	if (cudaStatus != hipSuccess) {
		mexErrMsgIdAndTxt("compdist:hipMalloc", "Can't Malloc dev_cmB!\n");
		goto Error;
	}
	// initialize output
    cudaStatus = hipMemset(dev_dist, 99.0, NA * NB * sizeof(double));
    if (cudaStatus != hipSuccess) {
		mexErrMsgIdAndTxt("compdist:hipMalloc", "Can't Memset dev_dist!\n");
		goto Error;
	}
	// transfer data
	cudaStatus = hipMemcpy(dev_A, A, NA * NA * sizeof(double), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		mexErrMsgIdAndTxt("compdist:hipMemcpyHostToDevice", "Can't transfer A to device!\n");
		goto Error;
	}
	cudaStatus = hipMemcpy(dev_B, B, NB * NB * sizeof(double), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		mexErrMsgIdAndTxt("compdist:hipMemcpyHostToDevice", "Can't transfer A to device!\n");
		goto Error;
	}
	cudaStatus = hipMemcpy(dev_idxA, idxA, NdCM * sizeof(double), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		mexErrMsgIdAndTxt("compdist:hipMemcpyHostToDevice", "Can't transfer cmA to device!\n");
		goto Error;
	}
	cudaStatus = hipMemcpy(dev_idxB, idxB, NdCM * sizeof(double), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		mexErrMsgIdAndTxt("compdist:hipMemcpyHostToDevice", "Can't transfer cmB to device!\n");
		goto Error;
	}
	cudaStatus = hipMemcpy(dev_dCM, dCM, NdCM * sizeof(double), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		mexErrMsgIdAndTxt("compdist:hipMemcpyHostToDevice", "Can't transfer cmB to device!\n");
		goto Error;
	}


	// launch kernel	

	compdistKernel<<<griddims, blockdims, dyn_sharedmemsize>>>(dev_A, dev_B, dev_idxA, dev_idxB, dev_dCM, NA, NB, NdCM, dev_dist);

	// check for errors
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		mexPrintf("hipGetLastError returned error code %d after launching compdistKernel!\n", cudaStatus);
		mexErrMsgIdAndTxt("compdist:hipGetLastError", "kernel didn't run correctly!\n");
		goto Error;
	}
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		mexPrintf("hipDeviceSynchronize returned error code %d after launching compdistKernel!\n", cudaStatus);
		mexErrMsgIdAndTxt("compdist:hipDeviceSynchronize", "device didn't sync!\n");
		goto Error;
	}

	// copy results to host
	cudaStatus = hipMemcpy(dist, dev_dist, NA * NB * sizeof(double), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		mexErrMsgIdAndTxt("compdist:hipMemcpyDeviceToHost", "couldn't get data back!\n");
		goto Error;
	}

	hipDeviceReset();
Error:
	hipDeviceReset();
	return cudaStatus;
}

__global__ void compdistKernel(const double* A, const double* B, 
								const double* idxA, const double* idxB, const double* dCM,
								const unsigned int NA, const unsigned int NB, const unsigned int NdCM,
								double* dist) {

	// prepare shared memory
	extern __shared__ float ins[]; // these memory blocks will be reused for different purposes.
	float *vA = ins; 
	float *vB = (float*) &vA[NdCM];

	// load indices of A and B
	int N_steps = (NdCM - 1) / blockDim.x + 1;// for large vectors, we load in steps
	for (int n = 0; n < N_steps; n++) {
		int idx_v = n*blockDim.x + threadIdx.x;
		if (idx_v < NdCM) {
			vA[idx_v] = __double2float_rn(idxA[idx_v]) - 1;
		}
	}
	__syncthreads();
	for (int n = 0; n < N_steps; n++) {
		int idx_v = n*blockDim.x + threadIdx.x;
		if (idx_v < NdCM) {
			vB[idx_v] = __double2float_rn(idxB[idx_v]) - 1;
		}
	}
	__syncthreads();

	// load values of A and B
	for (int n = 0; n < N_steps; n++) {
		int idx_v = n*blockDim.x + threadIdx.x;
		if (idx_v < NdCM) {
			vA[idx_v] = __double2float_rn(A[(int) (blockIdx.x*NA + vA[idx_v])]);
		}
	}
	__syncthreads();
	for (int n = 0; n < N_steps; n++) {
		int idx_v = n*blockDim.x + threadIdx.x;
		if (idx_v < NdCM) {
			vB[idx_v] = __double2float_rn(B[(int) (blockIdx.y*NB + vB[idx_v])]);
			// compute difference
			vA[idx_v] = powf(vA[idx_v] - vB[idx_v],2);
		}
	}
	__syncthreads();

	// load dCM and calculate integral
	for (int n = 0; n < N_steps; n++) {
		int idx_v = n*blockDim.x + threadIdx.x;
		if (idx_v < NdCM) {
			vA[idx_v] = vA[idx_v] * __double2float_rn(dCM[idx_v]);
		}
	}
	__syncthreads();

	// output final distance
	double dist_ij = (double) sum(vA, NdCM); 
	__syncthreads();
	if (threadIdx.x == 0) {
		dist[blockIdx.x*NB + blockIdx.y] = sqrtf(dist_ij);
	}
	__syncthreads();
}

// sum elements along x within a block
__device__ float sum(float* x, const int len) {
	int n_sums0 = lastPow2(len); // max number of cols in the initial sum step
	for (int n_sums = n_sums0; n_sums > 0; n_sums >>= 1) {
		int N_steps = (n_sums - 1) / blockDim.x + 1;
		for (int n = 0; n<N_steps; n++) {
			int idx = n*blockDim.x + threadIdx.x;
			if (idx < n_sums && (idx + n_sums) < len) {
				x[idx] += x[idx + n_sums];
			}
			__syncthreads();
		}
		__syncthreads();
	}
	__syncthreads();
	return x[0];
}

// the largest power of 2 smaller than input
__device__ int lastPow2(int n) {
	// next power of 2
	n--;
	n |= n >> 1;
	n |= n >> 2;
	n |= n >> 4;
	n |= n >> 8;
	n |= n >> 16;
	n++;
	// last power of 2
	return n >> 1;
}